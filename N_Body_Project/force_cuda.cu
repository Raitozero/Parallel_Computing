#include "hip/hip_runtime.h"
#include "Force.h"
#include "force_cuda.h"
#include <math.h>

__device__ float distance(const Particle& p, const Node& node) {
    return sqrt(pow((p.position.x - node.com.x), 2) + pow((p.position.y - node.com.y), 2));
}

__device__ Force computeForceHelper(const Particle& p1, const Particle& p2) {
    Force f = {0.0f, 0.0f};
    float dx = p2.position.x - p1.position.x;
    float dy = p2.position.y - p1.position.y;
    float distSquared = dx * dx + dy * dy;
    float dist = sqrt(distSquared);
    if (distSquared == 0) return f;
    float gravity = G * p1.mass * p2.mass / distSquared;
    f.x = gravity * dx / dist;
    f.y = gravity * dy / dist;
    return f;
}

__device__ Force computeForceFaraway(const Particle& p, const Node& node) {
    Force f = {0.0f, 0.0f};
    float dx = node.com.x - p.position.x;
    float dy = node.com.y - p.position.y;
    float distSquared = dx * dx + dy * dy;
    float dist = sqrt(distSquared);
    if (distSquared == 0) return f;
    float gravity = G * p.mass * node.mass / distSquared;
    f.x = gravity * dx / dist;
    f.y = gravity * dy / dist;
    return f;
}

__global__ void computeForces(Particle* particles, Force* forces, Node* nodes, int numParticles, float threshold) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= numParticles) return;

    Particle p = particles[idx];
    Force totalForce = {0.0f, 0.0f};

    // Traverse the quadtree and calculate forces
    for (int i = 0; i < numNodes; i++) {
        Node node = nodes[i];
        float s = node.length;
        float d = distance(p, node);

        if (s / d >= threshold) {
            if (node.isLeaf) {
                int particleIdx = node.particleIndex;
                Force f = computeForceHelper(p, particles[particleIdx]);
                totalForce.x += f.x;
                totalForce.y += f.y;
            } else {
                // Traverse children (handled by next iterations)
            }
        } else {
            Force f = computeForceFaraway(p, node);
            totalForce.x += f.x;
            totalForce.y += f.y;
        }
    }

    forces[idx] = totalForce;
}

void launchForceComputation(Particle* h_particles, Force* h_forces, Node* h_nodes, int numParticles, int numNodes, float threshold) {
    // Allocate device memory
    Particle* d_particles;
    Force* d_forces;
    Node* d_nodes;
    hipMalloc(&d_particles, numParticles * sizeof(Particle));
    hipMalloc(&d_forces, numParticles * sizeof(Force));
    hipMalloc(&d_nodes, numNodes * sizeof(Node));

    // Copy data from host to device
    hipMemcpy(d_particles, h_particles, numParticles * sizeof(Particle), hipMemcpyHostToDevice);
    hipMemcpy(d_nodes, h_nodes, numNodes * sizeof(Node), hipMemcpyHostToDevice);

    // Launch the kernel
    int blockSize = 256;
    int numBlocks = (numParticles + blockSize - 1) / blockSize;
    computeForces<<<numBlocks, blockSize>>>(d_particles, d_forces, d_nodes, numParticles, threshold);

    // Copy results back to host
    hipMemcpy(h_forces, d_forces, numParticles * sizeof(Force), hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_particles);
    hipFree(d_forces);
    hipFree(d_nodes);
}

vector<shared_ptr<Particle>> updateGenerateNew(const QuadTree& qt, const vector<shared_ptr<Particle>>& particles) {
    int numParticles = particles.size();
    vector<Particle> h_particles(numParticles);
    vector<Force> h_forces(numParticles);
    vector<Node> h_nodes; // Convert your quadtree to a flat array of nodes

    // Copy particles to the flat array
    for (int i = 0; i < numParticles; i++) {
        h_particles[i] = *particles[i];
    }

    // Launch CUDA kernel
    launchForceComputation(h_particles.data(), h_forces.data(), h_nodes.data(), numParticles, h_nodes.size(), threshold);

    // Update particle positions and velocities
    vector<shared_ptr<Particle>> newParticles(numParticles);
    for (int i = 0; i < numParticles; i++) {
        newParticles[i] = make_shared<Particle>(h_particles[i]);
        float accX = h_forces[i].x / h_particles[i].mass;
        float accY = h_forces[i].y / h_particles[i].mass;
        newParticles[i]->position.x += h_particles[i].velocity.x * unitTime + 0.5 * accX * unitTime * unitTime;
        newParticles[i]->position.y += h_particles[i].velocity.y * unitTime + 0.5 * accY * unitTime * unitTime;
        newParticles[i]->velocity.x += accX * unitTime;
        newParticles[i]->velocity.y += accY * unitTime;
    }

    return newParticles;
}
